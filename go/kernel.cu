#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ bool lucasLehmerTest(int p) {
    if (p == 2) return true;

    long long s = 4;
    long long m = (1LL << p) - 1;

    for (int i = 0; i < p - 2; i++) {
        s = (s * s - 2) % m;
    }

    return s == 0;
}

__global__ void lucasLehmerKernel(int *exponents, int *results, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        results[idx] = lucasLehmerTest(exponents[idx]) ? 1 : 0;
    }
}

extern "C" void launchLucasLehmer(int *exponents, int *results, int size) {
    int *d_exponents, *d_results;

    hipMalloc(&d_exponents, size * sizeof(int));
    hipMalloc(&d_results, size * sizeof(int));

    hipMemcpy(d_exponents, exponents, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    lucasLehmerKernel<<<blocksPerGrid, threadsPerBlock>>>(d_exponents, d_results, size);

    hipMemcpy(results, d_results, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_exponents);
    hipFree(d_results);
}
